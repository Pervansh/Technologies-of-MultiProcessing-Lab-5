#include <chrono>
#include <cstdio>
#include <filesystem>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"
#include ""

// #define USE_DOUBLE_PREC // Использование двойной точности в вычислениях вместо одинарной

#include "body.h"
#include "file.h"
#include "random.h"
#include "runge_kutta_cuda.cu"

std::filesystem::path PATH_TO_RESULTS = "results/random_m1";

int main(int argc, char **argv) {
    double t_0 = 0;
    double t_n = 2;
    double tau = 0.1;
    int steps = static_cast<int>((t_n - t_0) / tau);

    std::srand(static_cast<unsigned>(
        std::time(nullptr))); // Инициализация генератора случайных чисел

    // Список чисел тел, для которых выполняется программа
    std::vector<size_t> numbers_of_bodies = {100, 200, 300};

    // Цикл по количеству тел
    for (size_t n : numbers_of_bodies) {
        // Генерация случайных тел
        std::vector<Body> bodies = generate_random_bodies(n);

        size_t total_bodies = bodies.size();

        // Запуск метода Рунге-Кутты
        const double time_start = MPI_Wtime();
        runge_kutta_2_parallel_improved(
            bodies, bodies_local, t_0, t_n, tau, counts, offsets,
            PATH_TO_RESULTS / ("random_test_" + std::to_string(size) + "_" +
                               std::to_string(n) + ".txt"));
        const double time = MPI_Wtime() - time_start;

        if (rank == 0) {
            printf("processes: %d\nn: %zu\ntime: %f\nstep time: %f\n", size, n,
                   time, time / steps);
        }
    }

    // Освобождение ресурсов
    free_mpi_types();
    MPI_Finalize();

    return 0;
}
