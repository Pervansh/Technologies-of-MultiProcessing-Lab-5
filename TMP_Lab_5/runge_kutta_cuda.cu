#include "hip/hip_runtime.h"
#pragma once
#include <cstdio>
#include <string>
#include <vector>

#include <crt/hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""

#include "body.h"
#include "file.h"

__device__ void get_acceleration(const Body *bodies, int& n, const int& cudaBSshared, Vector3D *accelerations) {
    Vector3D difference;
    Vector3D acceleration;
    numType denom, difference_norm;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int localI = threadIdx.x;

    auto radius_vector_i = bodies[i].radius_vector;

    if (i < n) {
        extern __shared__ Vector3D sharedPoses[];
        extern __shared__ numType  sharedMasses[];

        numType res = 0;
        for (int bk = 0; bk < n; bk += cudaBSshared) {
            sharedPoses[localI]  = bodies[bk + localI].radius_vector;
            sharedMasses[localI] = bodies[bk + localI].mass;

            __syncthreads(); 

            for (int localK = 0; localK < cudaBSshared; ++localK) {
                difference = sharedPoses[localK] - radius_vector_i;
                difference_norm = difference.norm();
                denom = difference_norm * difference_norm * difference_norm;
                denom = denom > EPS ? denom : EPS;
                acceleration += difference * sharedMasses[localK] * (1. / denom);
            }
                // res += sharedPoses[localI * (cudaBSshared + 1) + localK] * b[localK * cudaBSshared + localJ];

            __syncthreads();
        }

        accelerations[i] = acceleration;
    }

    for (const auto &body : bodies) {
        difference = body.radius_vector - radius_vector_i;
        difference_norm = difference.norm();
        denom = difference_norm * difference_norm * difference_norm;
        denom = denom > EPS ? denom : EPS;
        acceleration += difference * body.mass * (1. / denom);
    }

    acceleration = acceleration * G;
}

// Параллельный метод Рунге-Кутты второго порядка
__host__ void runge_kutta_2_parallel(std::vector<Body> bodies,
                            std::vector<Body> bodies_local, double t_0,
                            double t_n, double tau,
                            const std::vector<int> &counts,
                            const std::vector<int> &offsets,
                            const std::string &filename) {
    std::ofstream file(filename);
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int local_size = counts[rank];

    std::vector<Body> bodies_local_buffer(bodies_local);
    std::vector<Vector3D> acceleration_prev;
    Vector3D acceleration_buffer, acceleration;

    for (double t = t_0; t < t_n + tau / 2.; t += tau) {
        // Процесс 0 записывает результаты в файл
        if (rank == 0) {
            for (size_t i = 0; i < bodies.size(); ++i) {
                write_step_to_file(t, bodies[i].radius_vector, filename);
            }
        }

        // Первый шаг метода Рунге-Кутты
        // printf("[INFO] Make first step from t = %f\n", t);
        for (size_t i = 0; i < local_size; ++i) {
            // printf("[INFO] Rank %d: \n", rank);
            get_acceleration(bodies, bodies_local[i].radius_vector,
                             acceleration_buffer);
            // printf("[INFO] Acceleration = {%f, %f, %f}\n",
            // acceleration_buffer.vector[0], acceleration_buffer.vector[1],
            // acceleration_buffer.vector[2]);

            bodies_local_buffer[i].radius_vector +=
                bodies_local[i].velocity * tau * 0.5;
            bodies_local_buffer[i].velocity += acceleration_buffer * tau * 0.5;

            // printf("[INFO] Rank %d: Body %zu: r = {%f, %f, %f}\n", rank, i +
            // 1, bodies_local_buffer[i].radius_vector.vector[0],
            // bodies_local_buffer[i].radius_vector.vector[1],
            // bodies_local_buffer[i].radius_vector.vector[2]);
        }

        // Синхронизируем данные между процессами
        MPI_Allgatherv(bodies_local_buffer.data(), local_size, mpi_body_type,
                       bodies.data(), counts.data(), offsets.data(),
                       mpi_body_type, MPI_COMM_WORLD);

        // printf("[INFO] Make second step from t = %f\n", t);
        for (size_t i = 0; i < local_size; ++i) {
            // printf("[INFO] Rank %d: \n", rank);
            get_acceleration(bodies, bodies_local_buffer[i].radius_vector,
                             acceleration);
            // printf("[INFO] Acceleration = {%f, %f, %f}\n",
            // acceleration.vector[0], acceleration.vector[1],
            // acceleration.vector[2]);
            bodies_local[i].radius_vector +=
                bodies_local_buffer[i].velocity * tau;
            bodies_local[i].velocity += acceleration * tau;

            // printf("[INFO] Rank %d: Body %zu: r = {%f, %f, %f}\n", rank, i +
            // 1, bodies_local[i].radius_vector.vector[0],
            // bodies_local[i].radius_vector.vector[1],
            // bodies_local[i].radius_vector.vector[2]);
        }

        // Синхронизируем данные между процессами
        MPI_Allgatherv(bodies_local.data(), local_size, mpi_body_type,
                       bodies.data(), counts.data(), offsets.data(),
                       mpi_body_type, MPI_COMM_WORLD);

        bodies_local_buffer = bodies_local;
    }

    // printf("[INFO] Rank %d: Finished solving system\n", rank);
}

// Параллельный метод Рунге-Кутты второго порядка
__host__ void runge_kutta_2_parallel_improved(std::vector<Body> bodies,
                                     std::vector<Body> bodies_local, double t_0,
                                     double t_n, double tau,
                                     const std::string &filename) {
    std::ofstream file(filename);

    std::vector<Body> bodies_local_buffer(bodies_local);
    std::vector<Vector3D> acceleration_prev;
    Vector3D acceleration_buffer, acceleration;

    for (double t = t_0; t < t_n + tau / 2.; t += tau) {
        // Процесс 0 записывает результаты в файл
        for (size_t i = 0; i < bodies.size(); ++i) {
            write_step_to_file(t, bodies[i].radius_vector, filename);
        }

        // Первый шаг метода Рунге-Кутты
        // printf("[INFO] Make first step from t = %f\n", t);
        for (size_t i = 0; i < local_size; ++i) {
            // printf("[INFO] Rank %d: \n", rank);
            get_acceleration(bodies, bodies_local[i].radius_vector,
                             acceleration_buffer);
            // printf("[INFO] Acceleration = {%f, %f, %f}\n",
            // acceleration_buffer.vector[0], acceleration_buffer.vector[1],
            // acceleration_buffer.vector[2]);

            bodies_local_buffer[i].radius_vector +=
                bodies_local[i].velocity * tau * 0.5;
            bodies_local_buffer[i].velocity += acceleration_buffer * tau * 0.5;

            // printf("[INFO] Rank %d: Body %zu: r = {%f, %f, %f}\n", rank, i +
            // 1, bodies_local_buffer[i].radius_vector.vector[0],
            // bodies_local_buffer[i].radius_vector.vector[1],
            // bodies_local_buffer[i].radius_vector.vector[2]);
        }

        // Синхронизируем данные между процессами
        MPI_Allgatherv(bodies_local_buffer.data(), local_size,
                       mpi_body_radius_vector_type_part, bodies.data(),
                       counts.data(), offsets.data(),
                       mpi_body_radius_vector_type_part, MPI_COMM_WORLD);

        // printf("[INFO] Make second step from t = %f\n", t);
        for (size_t i = 0; i < local_size; ++i) {
            // printf("[INFO] Rank %d: \n", rank);
            get_acceleration(bodies, bodies_local_buffer[i].radius_vector,
                             acceleration);
            // printf("[INFO] Acceleration = {%f, %f, %f}\n",
            // acceleration.vector[0], acceleration.vector[1],
            // acceleration.vector[2]);
            bodies_local[i].radius_vector +=
                bodies_local_buffer[i].velocity * tau;
            bodies_local[i].velocity += acceleration * tau;

            // printf("[INFO] Rank %d: Body %zu: r = {%f, %f, %f}\n", rank, i +
            // 1, bodies_local[i].radius_vector.vector[0],
            // bodies_local[i].radius_vector.vector[1],
            // bodies_local[i].radius_vector.vector[2]);
        }

        // Синхронизируем данные между процессами
        MPI_Allgatherv(bodies_local.data(), local_size,
                       mpi_body_radius_vector_type_part, bodies.data(),
                       counts.data(), offsets.data(),
                       mpi_body_radius_vector_type_part, MPI_COMM_WORLD);

        bodies_local_buffer = bodies_local;
    }

    // printf("[INFO] Rank %d: Finished solving system\n", rank);
}
